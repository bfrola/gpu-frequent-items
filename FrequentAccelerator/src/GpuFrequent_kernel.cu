#include "hip/hip_runtime.h"
/* ******************************************************
   ******************************************************
   
Frequent Items on GPU.
Sourceforge project: https://sourceforge.net/projects/figpu/

	This work is licensed under the Creative Commons
	Attribution-NonCommercial License. To view a copy of this license,
	visit http://creativecommons.org/licenses/by-nc/1.0/ or send a letter
	to Creative Commons, 559 Nathan Abbott Way, Stanford, California
	94305, USA.

	Algorithms and sources have been introduced and analysed in the following scientific paper: 
	*** U. Erra and B. Frola, In Proceedings of The International Conference on Computational Science (ICCS), Omaha, Nebraska, USA, June 4-6, 2012.
		http://www.sciencedirect.com/science/article/pii/S1877050912001317
	Please, cite this paper in your publication/work/project. Thank you.

   ******************************************************/

#pragma once

#include "GpuFrequent_kernel.h"

__global__ void kernel_Test1(int* data, int dataSize)
{
	int threadId = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (threadId >= dataSize)
		return;

	data[threadId]  = 1;
}

// -------------------------------------------------------------
// -------------------------------------------------------------

// 10/05/11
// No optimizations: Very slow 10x slower than optimized cpu
__global__ void kernel_Test2(int* inData, int* outData, int dataSize)
{
	int threadId = threadIdx.x + blockIdx.x * blockDim.x;
	
	int sum = 0;
	for (int i = 0; i < dataSize; i ++)
	{
		sum += inData[i];
	}

	//__syncthreads();

	outData[threadId]  = sum;
}

// -------------------------------------------------------------
// -------------------------------------------------------------

// Shared memory dynamically allocated 
extern __shared__ int sharedMemory[];

// cre 20/05/11
// mod 23/05/11
template <bool useSharedMemory>
__global__ void kernel_Test3(
	int* inData, // input data
	int2* outTData, // T data
	int dataSize, 
	int dataPerThread, 
	int outTSize)
{
	int threadId = threadIdx.x + blockIdx.x * blockDim.x;

	
	int* sh_inData;

	// Compilation-time condition
	if (useSharedMemory)
	{
		sh_inData = (int *) sharedMemory;

		// Load shared memory
		for (int i = 0; i < dataPerThread; i ++)
		{
			int offset = threadIdx.x * blockDim.x;
			sh_inData[offset + i] = inData[offset + i];
		}
	
		// Share data
		__syncthreads();
	}
	else
	{
		// Just copy the pointer
		sh_inData = inData;
	}
		
	// All threads load data
	// Only tSize threads execute the computation
	if (threadId > outTSize)
		return;

	int sum = 0;
	for (int i = 0; i < dataSize; i ++)
	{
		sum += sh_inData[i];
		//__syncthreads();
	}

	//printf("%d\n", blockIdx.x);
	outTData[threadId]  = make_int2(sum, 0);
}

// -------------------------------------------------------------
// -------------------------------------------------------------

// Uses float instead of int
template <bool useSharedMemory>
__global__ void kernel_Test4(
	float* inData, // input data
	int2* outTData, // T data
	int dataSize, 
	int dataPerThread, 
	int outTSize)
{
	int threadId = threadIdx.x + blockIdx.x * blockDim.x;
	
	float* sh_inData;

	// Compilation-time condition
	// Just copy the pointer
	sh_inData = inData;
		
	// All threads load data
	// Only tSize threads execute the computation
	if (threadId > outTSize)
		return;

	float sum = 0;
	for (int i = 0; i < dataSize; i ++)
	{
		sum += sh_inData[i];
	}

	outTData[threadId]  = make_int2((int) sum, 0);
}


// -------------------------------------------------------------
// -------------------------------------------------------------

// n = dataSize kernel calls
__global__ void kernel_Test5(
	int element, // input data
	int2* tData, // T data
	int tSize)
{
	int threadId = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (threadId > tSize)
		return;

	tData[threadId] = make_int2(element, element);
}

// TODO 9: Kernels version#1

// -------------------------------------------------------------
// -------------------------------------------------------------


// First attempt to write the frequent algo
__global__ void kernel_Test6(
	int* inData, // input data
	int inDataSize,  
	int2* tData, // T data
	int tSize)
{
	// UNUSED
}

// -------------------------------------------------------------
// -------------------------------------------------------------

// Copy from kernel 3
__global__ void kernel_Test7(
	int* inData, // input data
	int2* outTData, // T data
	int dataSize,  
	int outTSize, 
	int streamIndex)
{
	int threadId = threadIdx.x + blockIdx.x * blockDim.x;

	
	int* sh_inData;

		// Just copy the pointer
	sh_inData = inData;

		
	// All threads load data
	// Only tSize threads execute the computation
	if (threadId > outTSize)
		return;

	int sum = 0;
	for (int i = 0; i < dataSize; i ++)
	{
		sum += sh_inData[i];
		//__syncthreads();
	}

	//printf("%d\n", blockIdx.x);
	outTData[threadId]  = make_int2(sum, 0);
}

// -------------------------------------------------------------
// -------------------------------------------------------------

#define DEBUG_printfs 0

// Copy from kernel 7
// First GPU version of frequent
__global__ void kernel_Test8(
	int* inData, // input data
	int dataSize,  
	int dataOffset,
	int2* T, // T data, specif for this stream
	int TSize,
	int streamIndex)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	// All threads load data
	// Only tSize threads execute the computation
	if (idx >= TSize)
		return;

	__shared__ unsigned int counter; // atomicInc
	
	for (int i = dataOffset; i < dataSize + dataOffset; i ++)
	{
		//if (idx == 0)
		//	printf("[%d] --------------------\n", inData[i]);
		//printf("K%d\t %dx%d\n", idx, T[idx].x, T[idx].y);

		int found = (T[idx].x == inData[i]) ? 1 : 0;
		int foundInT = __syncthreads_or(found);

		//printf("K%d\t[%d] f: %d fInT: %d\n", idx, inData[i], found, foundInT);

		if (found)
		{
			T[idx].y ++;
		}

		if (foundInT)
		{
			continue;
		}
		
		// First half

		int isZero = (T[idx].y == 0) ? 1 : 0;
		int zerosInT = __syncthreads_or(isZero);
		
		
		if (isZero)
		{
			counter = 0;
			// Use atomic inc to compute isFirstZero

			unsigned int zeroRank = 0;

			zeroRank = atomicInc(&counter, blockDim.x); 
			// TOO SLOW! => sync the whole device
			// Solution: use reduction on shared mem
			if (zeroRank == 0) // Choose one of them
			{
				T[idx].x = inData[i];
				T[idx].y = 1;
			}
		}

		//printf("K%d\t[%d] z: %d zInT: %d zRank: %d\n", idx, inData[i], isZero, zerosInT, zeroRank);
		
		if (zerosInT)
		{
			continue;
		}

		T[idx].y --;
	} // for

} // Kernel test #8


// -------------------------------------------------------------
// -------------------------------------------------------------

// First version [T max size = blocksize]
// 2 __syncthreads_or, 1 atomic operation
__device__ void dfreq_2605(
	int idx,
	int dataItem, // input data
	int2& Titem) // T data, specif for this stream
{
	int found = (Titem.x == dataItem) ? 1 : 0;
	int foundInT = __syncthreads_or(found);

	if (found)
	{
		Titem.y ++;
	}

	if (foundInT)
	{
		return;
	}
		
	// First half

	int isZero = (Titem.y == 0) ? 1 : 0;
	int zerosInT = __syncthreads_or(isZero);

	__shared__ unsigned int counter;
		
	if (isZero)
	{
		counter = 0;
		// Use atomic inc to compute isFirstZero

		unsigned int zeroRank = 0;
		zeroRank = atomicInc(&counter, blockDim.x); 
		// VERY SLOW! => sync the whole device
		// Solution1: use reduction on shared mem
		// Solution2: use centralized approach
		
		if (zeroRank == 0) // Choose one of them
		{
			Titem.x = dataItem;
			Titem.y = 1;
		}
	}

	if (zerosInT)
	{
		return;
	}

	Titem.y --;
} // dfreq_2605


// ----------------------------------------------------------
// ----------------------------------------------------------

// Second version [T max size = blocksize]
// 2 syncthread, no atomic operations
// MULTIPLE THREADS WRITE THE SAME SH MEM LOCATION (selectedID)
// THIS DOES NOT WORK IN RELEASE MODE
__device__ void dfreq_3005(
	int idx,
	int dataItem, // input data
	int2& Titem) // T data, specif for this stream
{
	__shared__ bool foundInT;
	foundInT = false;
	
	// Sinse element of T are unique, only one thread at time can do this
	if (Titem.x == dataItem)
	{
		Titem.y ++;
		foundInT = true; 
	}

	// Share foundInT
	__syncthreads(); 
	
	// If one the element has been found
	if (foundInT)
	{
		return;
	}

	// ----------------------------------------
	// First half

	// Selectec the id of one of the items where y == 0
	__shared__ int selectedId;

	// By default is negative
	selectedId = -1;
	__syncthreads(); // Solve the false init problem

	// Set the current thread as candidate
	if (Titem.y == 0)
	{
		// Multiple threads write the same location
		// This does not work in RELEASE (no garantee on the result)
		selectedId = idx;
	}

	// Share selectedId
	__syncthreads();

	// Only one threads has been selected
	if (selectedId == idx) 
	{
		Titem.x = dataItem;
		Titem.y = 1;
	}
	
	// There are no zeros
	if (selectedId < 0) 
	{
		Titem.y --;
	}
} // dfreq_3005

// --------------------------------------------------------------------------
// --------------------------------------------------------------------------

// Equal to 3005, concorrent selectedId access resoved
__device__ void dfreq_0906(
	int idx,
	int dataItem, // input data
	int2& Titem) // T data, specif for this stream
{
	__shared__ bool foundInT;
	foundInT = false;
	__syncthreads(); // R2006 correct sharing of foundInT

	// Sinse element of T are unique, only one thread at time can do this
	if (Titem.x == dataItem)
	{
		Titem.y ++;
		foundInT = true; 
	}

	// Share foundInT
	__syncthreads(); 
	
	// If one the element has been found
	if (foundInT)
	{
		return;
	}

	// ----------------------------------------
	// First half

	// Select id of one item with Titem.y == 0
	__shared__ int selectedId; 
	selectedId = 0;
	__syncthreads(); // correct sharing of selectedId 

	// Set the current thread as candidate
	if (Titem.y == 0)
	{
		// 0 is NoValue
		selectedId = idx + 1;
	}

	// Share selectedId
	__syncthreads();

	// Only one threads has been selected
	if (selectedId == idx + 1) 
	{
		Titem.x = dataItem;
		Titem.y = 1;
	}
	
	// There are no zeros
	if (selectedId == 0) 
	{
		Titem.y --;
	}
} // dfreq_0906


// --------------------------------------------------------------------------
// --------------------------------------------------------------------------

// Copy from kernel 8
__global__ void kernel_Test9(
	int* inData, // input data
	int dataSize,  
	int dataOffset,
	int2* T, // T data, specif for this stream
	int TSize,
	int streamIndex)
{
	int idx = threadIdx.x; // Only blockIdx.x threads

	// Only tSize threads are active
	if (idx >= TSize)
		return;

	// Get the input (current state of T)
	int2 Titem = T[idx];

	for (int i = dataOffset; i < dataSize + dataOffset; i ++)
	{
		
		dfreq_0906(idx, inData[i], Titem);
		//dfreq_3005(idx, inData[i], Titem); // BUG in release mode
		//dfreq_2605(idx, inData[i], Titem); // Slow
		

#define DEBUG_printFrequentSteps 0
#if DEBUG_printFrequentSteps
		// DEBUG printfs
		if (idx == 0)
			printf("[%d] --------------------\n", inData[i]);
		printf("K%d\t %dx%d\n", idx, Titem.x, Titem.y);
#endif

	} // for

	// Write the output
	T[idx] = Titem;

} // Kernel test #9


__global__ void kernel_Test10(
	int* inData, // input data
	int dataSize,  
	int2* T, // T data, specif for this stream
	int TSize, 
	int TSizePow2)
{
	int idx = threadIdx.x; // More than blockIdx.x threads

	// Only tSize active threads 
	if (idx >= TSize)
		return;

	// Get the input (current state of T)
	int outputOffset = blockIdx.x * TSizePow2;
	int2 Titem = T[outputOffset + idx];

	
	// Offset
	int dataOffset = blockIdx.x * dataSize;
	int dataStart = dataOffset;
	int dataEnd = dataOffset + dataSize;

	for (int i = dataStart; i < dataEnd; i ++)
	{
		dfreq_0906(idx, inData[i], Titem); // same of T9

#define DEBUG_printFrequentSteps 0
#if DEBUG_printFrequentSteps
		// DEBUG printfs
		if (idx == 0)
			printf("\n[%d] --------------------\n", inData[i]);
		if (Titem.y > 0)
			printf("(K%d) %dx%d || ", idx, Titem.x, Titem.y);
#endif
	}

	// Write the output
	T[outputOffset + idx] = Titem;

} // Kernel test #10
