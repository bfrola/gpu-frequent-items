#include "hip/hip_runtime.h"
/* ******************************************************
/* ******************************************************

Frequent Items on GPU.
Sourceforge project: https://sourceforge.net/projects/figpu/

	This work is licensed under the Creative Commons
	Attribution-NonCommercial License. To view a copy of this license,
	visit http://creativecommons.org/licenses/by-nc/1.0/ or send a letter
	to Creative Commons, 559 Nathan Abbott Way, Stanford, California
	94305, USA.

	Algorithms and sources have been introduced and analysed in the following scientific paper: 
	*** U. Erra and B. Frola, In Proceedings of The International Conference on Computational Science (ICCS), Omaha, Nebraska, USA, June 4-6, 2012.
		http://www.sciencedirect.com/science/article/pii/S1877050912001317
	Please, cite this paper in your publication/work/project. Thank you.

/* ******************************************************
/* ******************************************************

Author: Bernardino Frola
Created: 19-jan-2011

 / *******************************************************/
#pragma once

#include "GPUFrequent.h"

// Sintax highlighting and Intellisense
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

// Help
#include "common/book.h"
#include "GPUFrequent_kernel.cu"

#include <stdlib.h>
#include <stdio.h>

using namespace BF;

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

GPUFrequent::GPUFrequent(float phi, int streamCount) :
m_StreamCount(streamCount)
{
	m_RunningTest = 10;

	m_K = (int) ceil (1.0/phi);

	// Smaller power of 2 > m_K
	m_KPow2 = (int)
			pow(2.0, ceil(log((double) m_K) / log(2.0) ));
	
	// ---------------------------------------------------------
	if (m_RunningTest >= 1 && m_RunningTest <= 5)
	{
		m_T.alloc(m_K);
	}
	// ---------------------------------------------------------
	else if (m_RunningTest == 8 || m_RunningTest == 9)
	{
		m_StreamPool = new hipStream_t[m_StreamCount];
		m_TPool = new GpuArray<int2>[m_StreamCount];

		for (int i = 0; i < m_StreamCount; i ++)
		{
			HANDLE_ERROR( hipStreamCreate( &m_StreamPool[i] ) );
			m_TPool[i].alloc(m_K);
		}
	}
	// ---------------------------------------------------------
	else if (m_RunningTest == 10)
	{
		m_T.alloc(m_KPow2 * m_StreamCount);
	}
	// ---------------------------------------------------------
	else
	{
		cout << "Invalid runnning test (" << m_RunningTest << ")" << endl;
		return;
	}


}
 

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

GPUFrequent::~GPUFrequent()
{
	// ---------------------------------------------------------
	if (m_RunningTest >= 1 && m_RunningTest <= 5)
	{
		m_T.free();
	}
	// ---------------------------------------------------------
	else if (m_RunningTest == 8 || m_RunningTest == 9)
	{
		for (int i = 0; i < m_StreamCount; i ++)
		{
			HANDLE_ERROR( hipStreamDestroy( m_StreamPool[i] ) );	
			m_TPool[i].free();
		}
	}
	// ---------------------------------------------------------
	else if (m_RunningTest == 10)
	{
		m_T.free();
	}
	// ---------------------------------------------------------
	else
	{
		cout << "Invalid runnning test (" << m_RunningTest << ")" << endl;
		return;
	}
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

void computeTheadsAndBlocks(
	int totalNumThreads, int blockSize, 
	dim3& threadBlock,
	dim3& blockGrid)
{
	threadBlock = dim3(blockSize, 1);
	blockGrid = dim3((int) ceil((double) totalNumThreads / threadBlock.x), 1);
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

void GPUFrequent::update_Test1to3( StripData* data, int dataSize, int dataOffset)
{
	StripDataVector<int>* intData = (StripDataVector<int>*) data; 

	int* hostData = &(*intData)[dataOffset];
	GpuArray<int> gaData;
	gaData.alloc(dataSize, false, false, false, hostData);
	gaData.copy(GpuArray<int>::HOST_TO_DEVICE);
	

	// Texture binding (works REALLY bad with integer values)
	//CUDA_SAFE_CALL(hipBindTexture(0, inDataCache,	
	//	gaData.getDevicePtr(), gaData.getSize() * sizeof(int)));
	
	int dataBlockSize = 4096; // blocks of X bytes

	dim3 threads, blocks;
	computeTheadsAndBlocks(m_KPow2, 512, threads, blocks);
	int dataSizePerThread = dataBlockSize / threads.x;

	int runs = ceil((double) dataSize / dataBlockSize);
	cout << "XLaunch: " << runs << " x " << dataBlockSize << endl;
	for (int i = 0; i < runs; i ++)
	{
		
		// TODO 9: Kernel call
		kernel_Test3<false><<<blocks, threads, dataBlockSize>>>(
			gaData.getDevicePtr(), 
			m_T.getDeviceWritePtr(), 
			dataBlockSize, 
			dataSizePerThread, 
			m_K);
	}

	// Unbind texture
	//CUDA_SAFE_CALL(hipUnbindTexture(inDataCache));
	m_T.copy(GpuArray<int2>::DEVICE_TO_HOST);

	
	// DO NOT FREE hostData
	gaData.free();
}

void GPUFrequent::update_Test5( StripData* data, int dataSize, int dataOffset)
{
	StripDataVector<int>* intData = (StripDataVector<int>*) data; 

	int* hostData = &(*intData)[dataOffset];
	GpuArray<int> gaData;
	gaData.alloc(dataSize, false, false, false, hostData);
	gaData.copy(GpuArray<int>::HOST_TO_DEVICE);

	dim3 threads, blocks;
	computeTheadsAndBlocks(m_KPow2, 512, threads, blocks);

	hipStream_t stream;
	HANDLE_ERROR( hipStreamCreate( &stream ) );

	for (int i = 0; i < 500; i ++)
	{
		kernel_Test5<<<blocks, threads, 0, stream>>>(
			1, m_T.getDeviceWritePtr(), m_K);
	}

	HANDLE_ERROR( hipStreamDestroy( stream ) );	

	gaData.free();
}

void GPUFrequent::update_Test6( StripData* data, int dataSize, int dataOffset)
{
	StripDataVector<int>* intData = (StripDataVector<int>*) data; 
	
	int* hostData = &(*intData)[dataOffset];
	GpuArray<int> gaData;
	gaData.alloc(dataSize, false, false, false, hostData);
	gaData.copy(GpuArray<int>::HOST_TO_DEVICE);
	
	int dataBlockSize = dataSize; // blocks of X bytes

	dim3 threads, blocks;
	computeTheadsAndBlocks(m_KPow2, 512, threads, blocks);
	
	kernel_Test6<<<blocks, threads>>>(
			gaData.getDevicePtr(), dataSize, 
			m_T.getDeviceWritePtr(), m_K);

	m_T.copy(GpuArray<int2>::DEVICE_TO_HOST);
	
	// DO NOT FREE hostData
	gaData.free();
}

void GPUFrequent::update_Test7_sub1(int streamCount, GpuArray<int> gaData, int dataSize)
{
	hipEvent_t timerStart, timerStop;
	hipEventCreate(&timerStart);
	hipEventCreate(&timerStop);

	hipEvent_t timerStart2, timerStop2;
	hipEventCreate(&timerStart2);
	hipEventCreate(&timerStop2);

	// ------------------------------	

	hipEventRecord(timerStart, 0);

	hipStream_t* stream;
	stream = new hipStream_t[streamCount];

	for (int i = 0; i < streamCount; i ++)
		HANDLE_ERROR( hipStreamCreate( &stream[i] ) );

	int dataBlockSize = dataSize / streamCount;

	dim3 threads, blocks;
	computeTheadsAndBlocks(m_KPow2, 1024, threads, blocks);

	// ------------------------------
	
	hipEventRecord(timerStart2, 0);
	
	for (int i = 0; i < streamCount; i ++)
	{
		kernel_Test7<<<blocks, threads, 0, stream[i]>>>(
			gaData.getDevicePtr(), 
			m_T.getDeviceWritePtr(), 
			dataBlockSize, 
			m_K, i);
	}

	// Wait for the end of any stream
	for (int i = 0; i < streamCount; i ++)
		HANDLE_ERROR( hipStreamSynchronize(stream[i]) );	

	hipEventRecord(timerStop2, 0);
	hipEventSynchronize(timerStop2);

	// ------------------------------

	for (int i = 0; i < streamCount; i ++)
		HANDLE_ERROR( hipStreamDestroy( stream[i] ) );	

	hipEventRecord(timerStop, 0);
	hipEventSynchronize(timerStop);

	float timerElapsed;
	hipEventElapsedTime(&timerElapsed, timerStart, timerStop);
	float speed = dataSize / timerElapsed;

	float timerElapsed2;
	hipEventElapsedTime(&timerElapsed2, timerStart2, timerStop2);
	float speed2 = dataSize / timerElapsed2;
	//cout << streamCount << " streams, speed: " << speed <<  " (" << timerElapsed << "ms)" << endl;
	cout << streamCount << " streams, speed: " << speed2 <<  " (" << timerElapsed2 << "ms)" << endl;

}

// 25/05/11 Multiple streams
void GPUFrequent::update_Test7( StripData* data, int dataSize, int dataOffset)
{
	hipEvent_t tStart, tInitStop, tAllocStop, tMemStop, tComputeStop, tFreeStop;
	hipEventCreate(&tStart);
	hipEventCreate(&tAllocStop);
	hipEventCreate(&tMemStop);
	hipEventCreate(&tComputeStop);
	hipEventCreate(&tFreeStop);

	// --------------------------------
	hipEventRecord(tStart, 0);
	// --------------------------------
	
	StripDataVector<int>* intData = (StripDataVector<int>*) data; 
	int* hostData = &(*intData)[dataOffset];
	GpuArray<int> gaData;
	gaData.alloc(dataSize, false, false, false, hostData);

	// --------------------------------
	hipEventRecord(tAllocStop, 0);
	hipEventSynchronize(tAllocStop);
	// --------------------------------

	gaData.copy(GpuArray<int>::HOST_TO_DEVICE);

	// --------------------------------
	hipEventRecord(tMemStop, 0);
	hipEventSynchronize(tMemStop);
	// --------------------------------

	// Test pool on m_StreamCount
	for (int i = 0; i < 92; i ++)
		update_Test7_sub1(i + 1, gaData, dataSize);
	
	// --------------------------------
	hipEventRecord(tComputeStop, 0);
	hipEventSynchronize(tComputeStop);
	// --------------------------------

	gaData.free();

	// --------------------------------
	hipEventRecord(tFreeStop, 0);
	hipEventSynchronize(tFreeStop);
	// --------------------------------

#define DEBUG_timings 0

	float elapsed;
#if DEBUG_timings
	hipEventElapsedTime(&elapsed, tStart, tAllocStop);
	cout << "Alloc: " << elapsed << "ms" << endl;
	hipEventElapsedTime(&elapsed, tAllocStop, tMemStop);
	cout << "Mem: " << elapsed << "ms" << endl;
	hipEventElapsedTime(&elapsed, tMemStop, tComputeStop);
	cout << "Com: " << elapsed << "ms" << endl;
	hipEventElapsedTime(&elapsed, tComputeStop, tFreeStop);
	cout << "Free: " << elapsed << "ms" << endl;
	hipEventElapsedTime(&elapsed, tStart, tFreeStop);
	cout << "Tot: " << elapsed << "ms" << endl;
#endif
	
	//m_T.copy(GpuArray<int2>::DEVICE_TO_HOST);

	
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

// 25/05/11 Multiple streams (copy form test7)
// Centralized stream init/destroy
// Updated 26/05
void GPUFrequent::update_Test8_9( StripData* data, int dataSize, int dataOffset)
{
	hipEvent_t tStart, tInitStop, tAllocStop, tMemStop, tComputeStop, tFreeStop;
	hipEventCreate(&tStart);
	hipEventCreate(&tAllocStop);
	hipEventCreate(&tMemStop);
	hipEventCreate(&tComputeStop);
	hipEventCreate(&tFreeStop);
	
	// --------------------------------
	hipEventRecord(tStart, 0);
	// --------------------------------
	
	StripDataVector<int>* intData = (StripDataVector<int>*) data; 
	int* hostData = &(*intData)[dataOffset];
	GpuArray<int> gaData;
	gaData.alloc(dataSize, false, false, false, hostData);
	
	// --------------------------------
	hipEventRecord(tAllocStop, 0);
	hipEventSynchronize(tAllocStop);
	// --------------------------------
	
	gaData.copy(GpuArray<int>::HOST_TO_DEVICE);
	
	dim3 threads, blocks;
	computeTheadsAndBlocks(m_KPow2, m_KPow2, threads, blocks);
	int dataBlockSize = dataSize / m_StreamCount;
	//cout << "XLaunch: " << threads.x << "x" << blocks.x << " S" << m_StreamCount << endl;
	 
	// --------------------------------
	hipEventRecord(tMemStop, 0);
	hipEventSynchronize(tMemStop);
	// --------------------------------
		
	for (int i = 0; i < m_StreamCount; i ++)
	{
		//kernel_Test8<<<blocks, threads, 0, m_StreamPool[i]>>>(
		kernel_Test9<<<blocks, threads, 0, m_StreamPool[i]>>>(
			gaData.getDevicePtr(), dataBlockSize, // Same input data
			i * dataBlockSize, // Data offset
			m_TPool[i].getDeviceWritePtr(), m_K, // many outputs (one for each stream)
			i // stream index
		);
	}
	
	for (int i = 0; i < m_StreamCount; i ++)
		HANDLE_ERROR( hipStreamSynchronize(m_StreamPool[i]) );
	
	// --------------------------------
	hipEventRecord(tComputeStop, 0);
	hipEventSynchronize(tComputeStop);
	// --------------------------------
	
	gaData.free();

	// --------------------------------
	hipEventRecord(tFreeStop, 0);
	hipEventSynchronize(tFreeStop);
	// --------------------------------

#define DEBUG_timings 0

	float elapsed;
	
#if DEBUG_timings
	//hipEventElapsedTime(&elapsed, tStart, tAllocStop);
	//cout << "Alloc: " << elapsed << "ms" << endl;
	//hipEventElapsedTime(&elapsed, tAllocStop, tMemStop);
	//cout << "Mem: " << elapsed << "ms" << endl;
	hipEventElapsedTime(&elapsed, tMemStop, tComputeStop);
	cout << "Com: " << elapsed << "ms" << endl;
	//hipEventElapsedTime(&elapsed, tComputeStop, tFreeStop);
	//cout << "Free: " << elapsed << "ms" << endl;
	//hipEventElapsedTime(&elapsed, tStart, tFreeStop);
	//cout << "Tot: " << elapsed << "ms" << endl;
#endif
} // Test #8

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

void GPUFrequent::update_Test10( StripData* data, int dataSize, int dataOffset)
{
	hipEvent_t tStart, tMemStop, tComputeStop;
	hipEventCreate(&tStart);
	hipEventCreate(&tMemStop);
	hipEventCreate(&tComputeStop);

	StripDataVector<int>* intData = (StripDataVector<int>*) data; 
	int* hostData = &(*intData)[dataOffset];
	GpuArray<int> gaData;
	gaData.alloc(dataSize, false, false, false, hostData);

	// --------------------------------
	hipEventRecord(tStart, 0);
	hipEventSynchronize(tStart);
	// --------------------------------
	
	gaData.copy(GpuArray<int>::HOST_TO_DEVICE);

	// --------------------------------
	hipEventRecord(tMemStop, 0);
	hipEventSynchronize(tMemStop);
	// --------------------------------

	dim3 threads, blocks;
	//computeTheadsAndBlocks(m_StreamCount * m_KPow2, m_KPow2, threads, blocks);
	threads.x = m_KPow2;
	blocks.x = m_StreamCount;
	
	int dataBlockSize = dataSize / m_StreamCount;
	
	kernel_Test10<<<blocks, threads>>>(
		gaData.getDevicePtr(), 
		dataBlockSize, 
		m_T.getDeviceWritePtr(), 
		m_K,		// Real size of T
		m_KPow2);	// Output offset on m_T

	// --------------------------------
	hipEventRecord(tComputeStop, 0);
	hipEventSynchronize(tComputeStop);
	// --------------------------------
	
#define DEBUG_timings 0
#if DEBUG_timings
	float elapsed;
	hipEventElapsedTime(&elapsed, tStart, tMemStop);
	cout << "Mem: " << elapsed << "ms" << endl;
	hipEventElapsedTime(&elapsed, tMemStop, tComputeStop);
	cout << "Com: " << elapsed << "ms" << endl;
	hipEventElapsedTime(&elapsed, tStart, tComputeStop);
	cout << "Tot: " << elapsed << "ms" << endl;
#endif

	gaData.free();
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------


#define DEBUG_INPUT_DATA 0
#define DEBUG_T_DATA 0

// Non device funcs
void GPUFrequent::update( StripData* data, int dataSize, int dataOffset)
{
	// TODO 9: temp limitation
	if (m_K > 1024)
	{
		cout << "GPUFrequent: K = " << m_K << " too large. Processing stopped." << endl;
		return;
	}

#if DEBUG_INPUT_DATA
	int sum = 0;
	for (int i = 0; i < 10; i ++)// TODO 6: Delete this
	{
		sum += hostData[i];
	}
	cout << "Sum: " << sum << endl;

	float sumF = 0.0f;
	for (int i = 0; i < 10; i ++)// TODO 6: Delete this
	{
		sumF += gaDataF.getHostPtr()[i];
	}
	cout << "Sum: " << sumF << endl; // Test OK
#endif

	// ---------
	//update_Test1to3(data, dataSize, dataOffset);
	//update_Test5(data, dataSize, dataOffset);
	//update_Test6(data, dataSize, dataOffset);
	//update_Test8_9(data, dataSize, dataOffset);

	update_Test10(data, dataSize, dataOffset);

	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	
	// ---------
#if DEBUG_T_DATA
	for (int i = 0; i < 3; i ++)// TODO 6: Delete this
	{
		//sum += hostData[i];
		cout << m_T.getHostPtr()[i].x << ", ";
	}
	cout << "..., " << m_T.getHostPtr()[m_T.getSize() - 1].x << endl;
#endif
	// ---------
}

OutputType GPUFrequent::getOutput(int thresh, int streamIndex)
{
	OutputType res;

	
	// -------------------------------------------------
	if (m_RunningTest == 8 || m_RunningTest == 9)
	{
		
		m_TPool[streamIndex].copy(GpuArray<int2>::DEVICE_TO_HOST);

		// DEBUG
		for (int j = 0; j < m_K; j ++)
		{
			//cout << "(" << m_TPool[streamIndex].getHostPtr()[j].x << ", " << m_TPool[streamIndex].getHostPtr()[j].y << ") ";
			if ( m_TPool[streamIndex].getHostPtr()[j].y > 0)
				res.insert(OutputItemPair(
					m_TPool[streamIndex].getHostPtr()[j].x, 
					m_TPool[streamIndex].getHostPtr()[j].y));
		}
	}
	// -------------------------------------------------
	else if (m_RunningTest == 10) 
	{
		// Require 
		int baseIdx = streamIndex * m_KPow2;
		
		//m_T.copy(GpuArray<int2>::DEVICE_TO_HOST, baseIdx, m_K);
		if (streamIndex == 0)
			m_T.copy(GpuArray<int2>::DEVICE_TO_HOST);
		
		for (int j = baseIdx; j < baseIdx + m_KPow2; j ++)
		{
			if ( m_T.getHostPtr()[j].y > 0)
				res.insert(OutputItemPair(
					m_T.getHostPtr()[j].x, 
					m_T.getHostPtr()[j].y));
		}
	}
	
	return res;
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

OutputType* GPUFrequent::getOutput(int thresh)
{
	OutputType* res = new OutputType[m_StreamCount];

	for (int i = 0; i < m_StreamCount; i ++)
	{
		res[i] = getOutput(thresh, i);
	}

	return res;
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

int GPUFrequent::getMemSize()
{
	// Gpu and cpu memory (2X)
	int singleStreamSize = m_KPow2 * sizeof(int2);
	return singleStreamSize * m_StreamCount;
}
