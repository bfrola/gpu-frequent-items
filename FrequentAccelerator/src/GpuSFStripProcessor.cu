/* ******************************************************
   ******************************************************
   
Frequent Items on GPU.
Sourceforge project: https://sourceforge.net/projects/figpu/

	This work is licensed under the Creative Commons
	Attribution-NonCommercial License. To view a copy of this license,
	visit http://creativecommons.org/licenses/by-nc/1.0/ or send a letter
	to Creative Commons, 559 Nathan Abbott Way, Stanford, California
	94305, USA.

	Algorithms and sources have been introduced and analysed in the following scientific paper: 
	*** U. Erra and B. Frola, In Proceedings of The International Conference on Computational Science (ICCS), Omaha, Nebraska, USA, June 4-6, 2012.
		http://www.sciencedirect.com/science/article/pii/S1877050912001317
	Please, cite this paper in your publication/work/project. Thank you.

   ******************************************************/
/* *****************************************************

Author: Bernardino Frola
Created: 30-jun-2011

/ *******************************************************/

#include "GpuSFStripProcessor.h"
#include <algorithm>
#include <numeric>
#include <math.h>
#include <iomanip>
#include <thrust/sort.h>
#include <thrust/unique.h>
# include <thrust/iterator/constant_iterator.h>
#include <thrust/binary_search.h>

#include "hip/hip_vector_types.h"

#include "Utils.h"


struct c_unique {
  int current;
  c_unique() {current=0;}
  int operator()() {return current++;}
} UniqueNumber2;


// DEBUG REDUCE
device_T<int> tempT;

// TEMP for binary search
thrust::device_vector<int> bucket_begin;
thrust::device_vector<int> bucket_end;
thrust::device_vector<int> bucket_size;
thrust::device_vector<int> d_vec_unique;


GpuSFStripProcessor::GpuSFStripProcessor (float phi, int stripSize) : SF(phi, stripSize)
{
	m_T.resize(m_TSize); // Fixed
	cout << "GpuSF::TSize = " << m_T.size() << endl;
	m_EnableNewEntriesAdv = false;

	tempT.resize(m_TSize);

	bucket_begin.resize(m_TSize);
	bucket_end.resize(m_TSize);
	d_vec_unique.resize(m_TSize);
	bucket_size.resize(m_TSize);
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

GpuSFStripProcessor::~GpuSFStripProcessor()
{
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------


struct incFunc
{
	int m_IncVal;
	incFunc(int incVal)
	{
		m_IncVal = incVal;
	}

    __host__ __device__
    int operator()(int current)
    {
        return current + m_IncVal;
    }
};

struct computeCounter
{
	int* m_Items;
	int* m_Counters;

	computeCounter(int* itemsRef, int* countersRef)
	{
		m_Items = itemsRef;
		m_Counters = countersRef;
	}

    __host__ __device__
    int operator()(int x, int y)
    {
		if (m_Items[y] == 0)
			return 0;

		int diff = x - y - 1;	// Difference ignoring the first 1
		diff += m_Counters[y];	// Add the counter value instead of the first 1
        return diff;
    }
};


// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

void GpuSFStripProcessor::fillStrip(StripDataVector<int>* intData, int scOffset, int scSize)
{
	// Get the pointer to data
	int* subData = &(*intData)[scOffset];

	thrust::copy(subData, subData + scSize, m_T.items().end() - scSize - 1);

	// Set counters of new elemnents to 1
	thrust::fill(m_T.counters().end() - scSize - 1,	m_T.counters().end() - 1, 1);

	// Last element of T set to (0, 0)
	m_T.set(m_T.size() - 1, 0, 0); 
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

int GpuSFStripProcessor::getIncDecValue()
{
	return m_T.counters()[m_K - 1];
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------


void GpuSFStripProcessor::incrementCounters(int value)
{
	thrust::transform(m_T.counters().begin(), m_T.counters().begin() + m_K, m_T.counters().begin(), incFunc(value));
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------


void GpuSFStripProcessor::sortStripByItems()
{
	thrust::sort_by_key(m_T.items().begin(), m_T.items().end(), m_T.counters().begin()); // sort order is irrilevant
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------


void GpuSFStripProcessor::reduceStrip()
{
	// Binary search instead of reduce_by_key
	// See svn for old versions

	thrust::device_vector<int>::iterator d_vec_unique_end = 
		thrust::unique_copy(m_T.items().begin(), m_T.items().end(), d_vec_unique.begin());

	thrust::counting_iterator<int> search_begin(0);

	thrust::lower_bound(m_T.items().begin(), m_T.items().end(),
		d_vec_unique.begin(), d_vec_unique_end, 
		bucket_begin.begin());

	thrust::upper_bound(m_T.items().begin(), m_T.items().end(),
		d_vec_unique.begin(), d_vec_unique_end, 
		bucket_end.begin());

	thrust::transform(bucket_end.begin(), bucket_end.end(),
		bucket_begin.begin(), 
		bucket_size.begin(), 
		computeCounter(
			thrust::raw_pointer_cast(&m_T.items()[0]),
			thrust::raw_pointer_cast(&m_T.counters()[0])));

	thrust::copy(d_vec_unique.begin(), d_vec_unique_end, m_T.items().begin());
	thrust::copy(bucket_size.begin(), bucket_size.end(), m_T.counters().begin());

	int newUniqueSize = d_vec_unique_end - d_vec_unique.begin();
	int remaining = m_T.size() - newUniqueSize;

	thrust::fill(m_T.items().end() - remaining, m_T.items().end(), 0);
	thrust::fill(m_T.counters().end() - remaining, m_T.counters().end(), 0);
}

void GpuSFStripProcessor::reduceStripOV()
{
	thrust::pair<thrust::device_vector<int>::iterator, 
		thrust::device_vector<int>::iterator> newEnd = 
		thrust::reduce_by_key(
			m_T.items().begin(), m_T.items().end(), m_T.counters().begin(), 
			m_T.items().begin(), m_T.counters().begin());

	// Clean old values
	thrust::fill(newEnd.first, m_T.items().end(), 0);
	thrust::fill(newEnd.second, m_T.counters().end(), 0);
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

		
void GpuSFStripProcessor::sortStripByCounters()
{
	thrust::sort_by_key(m_T.counters().begin(), m_T.counters().end(), m_T.items().begin(), thrust::greater<int>());
}


// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

// Print information on substrip size without affecting performances measurements
void GpuSFStripProcessor::printSubStripInfo(StripData* data)
{
	StripDataVector<int>* intData = (StripDataVector<int>*) data; 
	int stripSize = (m_StripSize > 0) ? m_StripSize : intData->size();

	int subStripSize = getStripSize();	
	int subStripCount = stripSize / subStripSize;
	int remaining = stripSize - subStripCount * subStripSize;

	cout << "(Gpu) Data: " << stripSize << " m_K: " << m_K << 
		" |T|: " << m_T.size() <<
		" subStrip: " << subStripCount << "x" << subStripSize <<
		"+" << remaining << 
		endl;
}

void GpuSFStripProcessor::printT(string caption, int sizeLimit)
{
	m_T.print(caption, sizeLimit);
}

	
// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

/*
// FOR DEBUGGING
void GpuSFStripProcessor::processSubStrip(StripDataVector<int>* intData, int scOffset, int scSize)
{
	cout << "*************************************************************************" << endl;
	cout << "From" << scOffset << " x " << scSize << endl;

	MultiTimer<false> mTimer; 
	mTimer.eventRecord("Start");
	
	// Get the pointer to data
	int* subData = &(*intData)[scOffset];



	thrust::copy(subData, subData + scSize, m_T.items().end() - scSize - 1);

	mTimer.eventRecord("Copy");

	// Set counters of new elemnents to 1
	thrust::fill(m_T.counters().end() - scSize - 1,	m_T.counters().end() - 1, 1);

	// Last element of T set to (0, 0)
	m_T.set(m_T.size() - 1, 0, 0); 

	int minCounter = m_T.counters()[m_K - 1];
	bool enableNewEntryAdv = m_EnableNewEntriesAdv && (minCounter > 0);
	
	int printTsize = 0;

	if (enableNewEntryAdv)
	{
		// First K counters of T decremented by minCounter
		thrust::transform(m_T.counters().begin(), m_T.counters().begin() + m_K, m_T.counters().begin(), incFunc(-minCounter));

		//int * countersRawPointer = thrust::raw_pointer_cast(&m_T.counters()[0]);
		// use ptr in a CUDA C kernel
		//incrementKernel<<<(m_K / 256) + 1, 256>>>(countersRawPointer, m_K, -minCounter);
		
		//m_T.print("DEC: after <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<< ", printTsize);

		mTimer.eventRecord("Decr");
	}
	

	// Sort by items -> items and counters in m_TItems
	thrust::sort_by_key(m_T.items().begin(), m_T.items().end(), m_T.counters().begin());

	m_T.print("SORT <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<< ", printTsize);

	mTimer.eventRecord("Sort1");

	int position = 2085;

	cout << "BEFORE" << endl;
	cout << "\tItem[" << position << "] = " << m_T.items()[position] << endl;
	cout << "\tCoun[" << position << "] = " << m_T.counters()[position] << endl;


	thrust::device_vector<int>::iterator d_vec_unique_end = 
		thrust::unique_copy(m_T.items().begin(), m_T.items().end(), d_vec_unique.begin());

	thrust::counting_iterator<int> search_begin(0);

	thrust::lower_bound(m_T.items().begin(), m_T.items().end(),
		d_vec_unique.begin(), d_vec_unique_end, 
		bucket_begin.begin());

	thrust::upper_bound(m_T.items().begin(), m_T.items().end(),
		d_vec_unique.begin(), d_vec_unique_end, 
		bucket_end.begin());

	//thrust::fill(bucket_size.begin(), bucket_size.end(), 0);

	cout << "bucket_end: end() - begin() = " << (bucket_end.end() - bucket_end.begin()) << endl;

	thrust::transform(bucket_end.begin(), bucket_end.end(),
		bucket_begin.begin(), 
		bucket_size.begin(), 
		computeCounter(
			thrust::raw_pointer_cast(&m_T.items()[0]),
			thrust::raw_pointer_cast(&m_T.counters()[0])));

	cout << "AFTER1" << endl;
	cout << "\tItem[" << position << "] = " << m_T.items()[position] << endl;
	cout << "\tCoun[" << position << "] = " << m_T.counters()[position] << endl;

	mTimer.eventRecord("Reduce");	
		
	thrust::copy(d_vec_unique.begin(), d_vec_unique_end, m_T.items().begin());
	thrust::copy(bucket_size.begin(), bucket_size.end(), m_T.counters().begin());

	cout << "AFTER2" << endl;
	cout << "\tItem[" << position << "] = " << m_T.items()[position] << endl;
	cout << "\tCoun[" << position << "] = " << m_T.counters()[position] << endl;

	int newUniqueSize = d_vec_unique_end - d_vec_unique.begin();
	int remaining = m_T.size() - newUniqueSize;

	thrust::fill(m_T.items().end() - remaining, m_T.items().end(), 0);
	thrust::fill(m_T.counters().end() - remaining, m_T.counters().end(), 0);

	m_T.print("REDUCE <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<< ", printTsize);

	mTimer.eventRecord("Fill");

	// Sort by counters -> items and counters
	thrust::sort_by_key(m_T.counters().begin(), m_T.counters().end(), m_T.items().begin(), thrust::greater<int>());

	//m_T.print("SORT2 <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<< ", printTsize);

	mTimer.eventRecord("Sort2");
	
	if (enableNewEntryAdv)
	{
		// First K counters of T incremented by minCounter: an advatage to new entries
		thrust::transform(m_T.counters().begin(), m_T.counters().begin() + m_K, m_T.counters().begin(), incFunc(minCounter));

		//int * countersRawPointer = thrust::raw_pointer_cast(&m_T.counters()[0]);
		// use ptr in a CUDA C kernel
		//incrementKernel<<<(m_K / 256) + 1, 256>>>(countersRawPointer, m_K, minCounter);
		
		//m_T.print("INC: after <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<< ", printTsize);

		mTimer.eventRecord("Incr");
	}

	m_T.print("OUT <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<");
	
	mTimer.printTimes();	
}
*/

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

OutputType GpuSFStripProcessor::getOutput(int thresh)
{
	OutputType res;

	if (m_T.items().size() < m_K)
	{
		return res;
	}

	thrust::host_vector<int> h_items(m_K);
	thrust::copy(m_T.items().begin(), m_T.items().begin() + m_K, 
		h_items.begin());
	
	thrust::host_vector<int> h_counters(m_K);
	thrust::copy(m_T.counters().begin(), m_T.counters().begin() + m_K, 
		h_counters.begin());
	
	//thresh = 0;

	for (int i = 0; i < m_K; i ++)
	{
		int count = h_counters[i];

		if (count >= thresh)
			res.insert(OutputItemPair(h_items[i], count));
	}

	return res;
}

int GpuSFStripProcessor::getSize()
{
	return m_T.size() * sizeof(int) * 6; // items, counters, temp data for binary search (4 arrays)
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------
// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

template <class Type>
void device_T<Type>::resize(int size)
{
	m_Size = size;
	items().resize(m_Size);
	counters().resize(m_Size);
	//h_items().resize(m_Size);
	//h_counters().resize(m_Size);
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

template <class Type>
string device_T<Type>::toStream(string caption, int sizeLimit = 0)
{
	stringstream s;
	s << caption.c_str() << endl;

	if (sizeLimit == 0)
		sizeLimit = size();

	thrust::host_vector<int> h_items(sizeLimit);
	thrust::copy(items().begin(), items().begin() + sizeLimit, h_items.begin());
	
	thrust::host_vector<int> h_counters(sizeLimit);
	thrust::copy(counters().begin(), counters().begin() + sizeLimit, h_counters.begin());

	for (int i = 0; i < sizeLimit; i ++)
	{
		s << i << ") " << h_items[i]  << ", " << h_counters[i] << endl;
	}
	return s.str();
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

template <class Type>
void device_T<Type>::print(string caption, int sizeLimit) 
{
	ofstream fileStream;
	fileStream.open ("example_gpu.dat", ios::app);
	//cout << toStream(caption).c_str();
	fileStream << toStream(caption, sizeLimit).c_str();
	fileStream.close();
	//cout << toStream(caption).c_str();
}

// -------------------------------------------------------------------------
// -------------------------------------------------------------------------

template <class Type>
void device_T<Type>::set(int pos, Type item, Type counter)
{
	items()[pos] = item;
	counters()[pos] = counter;

	//h_items()[pos] = item;
	//h_counters()[pos] = counter;
}

template class device_T<int>;
