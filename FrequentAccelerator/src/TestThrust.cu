#include "hip/hip_runtime.h"
/* ******************************************************
   ******************************************************
   
Frequent Items on GPU.
Sourceforge project: https://sourceforge.net/projects/figpu/

	This work is licensed under the Creative Commons
	Attribution-NonCommercial License. To view a copy of this license,
	visit http://creativecommons.org/licenses/by-nc/1.0/ or send a letter
	to Creative Commons, 559 Nathan Abbott Way, Stanford, California
	94305, USA.

	Algorithms and sources have been introduced and analysed in the following scientific paper: 
	*** U. Erra and B. Frola, In Proceedings of The International Conference on Computational Science (ICCS), Omaha, Nebraska, USA, June 4-6, 2012.
		http://www.sciencedirect.com/science/article/pii/S1877050912001317
	Please, cite this paper in your publication/work/project. Thank you.

   ******************************************************
   ******************************************************/

#include "TestThrust.h"

# include <thrust/host_vector.h>
# include <thrust/device_vector.h>
# include <thrust/sort.h>
# include <thrust/unique.h>
# include <thrust/iterator/constant_iterator.h>
#include <thrust/binary_search.h>

#include "Utils.h"

// return a random float2 in [0,1)^2float

struct randGenerator
{
	int multiply;
	randGenerator(int m)
	{
		multiply = m;
	}

	int operator()()
    {
		int val = multiply * rand() / RAND_MAX;
		//cout << val << endl;
        return val;
    }
};

int make_myrand2(void){
	return 1000000 * rand() / RAND_MAX;
}

void testThrust(const int size)
{
	// generate 16M random numbers on the host
	//thrust::host_vector<int> h_vec(1 << 24);
	//const int size = 1 << 20;

	cout << "Size: " << size << endl;

	thrust::host_vector<int> h_vec(size);
	thrust::host_vector<int> h_conuter(size);


	thrust::generate(h_vec.begin(), h_vec.end(), make_myrand2);
	cout << "rand generated" << endl;

	// transfer data to the device
	
	thrust::device_vector<int> d_vec= h_vec;
	thrust :: device_vector <int> d_counter (size);
	thrust::device_vector<int> d_vec_out(size);

	hipEvent_t tStart, tSort, tReduce, tSearch;
	hipEventCreate(&tStart);
	hipEventCreate(&tSort);
	hipEventCreate(&tReduce);
	hipEventCreate(&tSearch);

	hipEventRecord(tStart, 0);
	hipEventSynchronize(tStart);
	
	// sort data on the device (805 Mkeys/sec on GeForce GTX 480)
	thrust::sort(d_vec.begin(), d_vec.end());

	hipEventRecord(tSort, 0);
	hipEventSynchronize(tSort);

	// allocate some random points in the unit square on the host
	thrust::reduce_by_key(d_vec.begin(), d_vec.end(),
		thrust::make_constant_iterator(1),
		d_vec_out.begin(),
		d_counter.begin());

	hipEventRecord(tReduce, 0);
	hipEventSynchronize(tReduce);

	
	// bucket_begin[i] indexes the first element of bucket i
	// bucket_end[i] indexes one past the last element of bucket i
	thrust::device_vector<int> bucket_begin(size);
	thrust::device_vector<int> bucket_end(size);
	// used to produce integers in the range [0, w*h)
	thrust::counting_iterator<int> search_begin(0);
	// find the beginning of each bucket's list of points
	thrust::lower_bound(d_vec.begin(), d_vec.end(),
		search_begin, search_begin + size, bucket_begin.begin());
	// find the end of each bucket's list of points
	thrust::upper_bound(d_vec.begin(), d_vec.end(),
		search_begin, search_begin + size, bucket_end.begin());

	// allocate space to hold per-bucket sizes
	thrust::device_vector<unsigned int> bucket_sizes(size);

	// take the difference between bounds to find each bucket�s size
	thrust::transform(
		bucket_end.begin(), bucket_end.end(),
		bucket_begin.begin(),
		bucket_sizes.begin(),
		minus<unsigned int>());
	// bucket_size[i] = bucket_end[i] �bucket_begin[i]
	// note each bucket�s size appears in the output even if it is empty

	hipEventRecord(tSearch, 0);
	hipEventSynchronize(tSearch);

	// transfer data back to host
	thrust::copy(d_counter.begin(), d_counter.end(), h_conuter.begin());
	thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
	
	cout << "D2H" << endl;

	float tElapsed;
	hipEventElapsedTime(&tElapsed, tStart, tSort);
	cout << "sort:\t" << tElapsed << "ms\t(" << (size / tElapsed) << ")" << endl;
	hipEventElapsedTime(&tElapsed, tSort, tReduce);
	cout << "reduce:\t" << tElapsed << "ms\t(" << (size / tElapsed) << ")" << endl;
	hipEventElapsedTime(&tElapsed, tReduce, tSearch);
	cout << "Search:\t" << tElapsed << "ms\t(" << (size / tElapsed) << ")" << endl;
	hipEventElapsedTime(&tElapsed, tStart, tSearch);
	cout << "Total:\t" << tElapsed << "ms\t(" << (size / tElapsed) << ")" << endl;


	//int maxSize = 150;
	//int lastElem = (h_vec.size() < maxSize) ? h_vec.size() : maxSize;
	//for ( int i = 0; i < maxSize ; i ++)
	//	std :: cout << i << " [" << h_vec[i] << ", " << h_conuter[i] << "]" << endl ;
	// H and D are automatically deleted when the function returns

	system("pause");
}

float make_myrandf(void){
	return 1000000 * rand() / RAND_MAX;
}

void testThrustBinarySearch(const int size, const int domainSize)
{
	MultiTimer<true> mTimer; 
	
	cout << "Size: " << size << endl;

	thrust::host_vector<float> h_vec(size);
	thrust::host_vector<float> h_conuter(size);

	thrust::device_vector<int> bucket_begin(size);
	thrust::device_vector<int> bucket_end(size);
	thrust::device_vector<int> d_vec_unique(size);

	thrust::generate(h_vec.begin(), h_vec.end(), randGenerator(domainSize));
	cout << "rand generated" << endl;

	//thrust::sequence(h_conuter.begin(), h_conuter.end());
	//cout << "sequence generated" << endl;

	//mTimer.eventRecord("Rand generation");
	mTimer.eventRecord("Start");

	thrust::device_vector<float> d_vec= h_vec;
	//thrust :: device_vector <float> d_counter = h_conuter;
	thrust::device_vector <float> d_counter(size);
	thrust::fill(d_counter.begin(), d_counter.end(), 1);

	mTimer.eventRecord("Copy H2D");

	//thrust::sort(d_vec.begin(), d_vec.end());
	//thrust::sort_by_key(d_vec.begin(), d_vec.end(), d_counter.begin());
	thrust::stable_sort_by_key(d_vec.begin(), d_vec.end(), d_counter.begin());

	mTimer.eventRecord("Sort by items");

	/*
	thrust::reduce_by_key(d_vec.begin(), d_vec.end(),
		d_counter.begin(),
		d_vec.begin(),
		d_counter.begin());
	*/

	thrust::device_vector<int>::iterator d_vec_unique_end = 
		thrust::unique_copy(d_vec.begin(), d_vec.end(), d_vec_unique.begin());

	thrust::counting_iterator<int> search_begin(0);

	thrust::lower_bound(d_vec.begin(), d_vec.end(),
		d_vec_unique.begin(), d_vec_unique_end, 
		bucket_begin.begin());

	thrust::upper_bound(d_vec.begin(), d_vec.end(),
		d_vec_unique.begin(), d_vec_unique_end, 
		bucket_end.begin());

	thrust::transform(bucket_end.begin(), bucket_end.end(),
		bucket_begin.begin(), 
		d_counter.begin(), 
		minus<int>());

	thrust::copy(d_vec_unique.begin(), d_vec_unique_end, d_vec.begin());

	int newUniqueSize = d_vec_unique_end - d_vec_unique.begin();
	thrust::fill(d_vec_unique.end() - newUniqueSize, d_vec_unique.end(), 0);

	mTimer.eventRecord("Reduce");


	thrust::sort_by_key(d_counter.begin(), d_counter.end(), d_vec.begin(), thrust::greater<int>());

	mTimer.eventRecord("Sort by counters");

	// transfer data back to host
	//thrust::copy(d_counter.begin(), d_counter.end(), h_conuter.begin());
	//thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
	
	//mTimer.eventRecord("Copy D2H");

	mTimer.printTimes();

	cout << endl;

	for (int i = 0; i < 100; i++)
	{
		if (i == d_vec_unique.size())
			break;

		cout << i << ") " << d_vec[i] << " x " << d_counter[i] << endl;
	}

	system("pause");
}